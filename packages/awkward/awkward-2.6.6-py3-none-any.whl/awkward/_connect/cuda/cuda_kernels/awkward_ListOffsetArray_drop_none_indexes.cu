#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (tooffsets, noneindexes, fromoffsets, length_offsets, length_indexes, invocation_index, err_code) = args
//     scan_in_array = cupy.zeros(length_indexes, dtype=cupy.int64)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_ListOffsetArray_drop_none_indexes_a", tooffsets.dtype, noneindexes.dtype, fromoffsets.dtype]))(grid, block, (tooffsets, noneindexes, fromoffsets, length_offsets, length_indexes, scan_in_array, invocation_index, err_code))
//     scan_in_array = cupy.cumsum(scan_in_array)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_ListOffsetArray_drop_none_indexes_b", tooffsets.dtype, noneindexes.dtype, fromoffsets.dtype]))(grid, block, (tooffsets, noneindexes, fromoffsets, length_offsets, length_indexes, scan_in_array, invocation_index, err_code))
// out["awkward_ListOffsetArray_drop_none_indexes_a", {dtype_specializations}] = None
// out["awkward_ListOffsetArray_drop_none_indexes_b", {dtype_specializations}] = None
// END PYTHON

template <typename T, typename C, typename U>
__global__ void
awkward_ListOffsetArray_drop_none_indexes_a(
    T* tooffsets,
    const C* noneindexes,
    const U* fromoffsets,
    int64_t length_offsets,
    int64_t length_indexes,
    int64_t* scan_in_array,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t offset1 = 0;

    if (thread_id < length_offsets) {
      if (thread_id > 0) {
        int64_t offset1 = fromoffsets[thread_id - 1];
      }
      int64_t offset2 = fromoffsets[thread_id];
      for (int j = offset1 + threadIdx.y; j < offset2; j += blockDim.y) {
        if (noneindexes[j] < 0) {
          scan_in_array[j] = 1;
        }
      }
    }
  }
}

template <typename T, typename C, typename U>
__global__ void
awkward_ListOffsetArray_drop_none_indexes_b(
    T* tooffsets,
    const C* noneindexes,
    const U* fromoffsets,
    int64_t length_offsets,
    int64_t length_indexes,
    int64_t* scan_in_array,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < length_offsets) {
      int64_t nr_of_nones = thread_id > 0 ? scan_in_array[fromoffsets[thread_id] - 1] : 0;
      tooffsets[thread_id] = fromoffsets[thread_id] - nr_of_nones;
    }
  }
}
